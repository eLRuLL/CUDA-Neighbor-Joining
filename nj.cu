
#include <hip/hip_runtime.h>
#include <math.h>
#include <fstream>
#include <stdio.h>
#include <exception>

#define MAX_THREADS 1024

using namespace std;

__global__ void nj_step1(float* mat, float* res,int width)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < width){
		float rpta = 0.0f;
		for(int i=0; i<width; i++)
		{
			if(i<idx)
				rpta += mat[idx*width + i];
			else
				rpta += mat[i*width + idx];
		}
		res[idx] = rpta;
	}
}

__global__ void nj_step2(float* mat, float* mat2, float* diverg, int width, int* fronteras)
{
	int bx = blockIdx.x;
	
	int i = 0;
	int currfil = 0;
	int currcol = 0;
	while(fronteras)
	{
		if(bx < fronteras[i])
			currfil = 1;
			currcol = fronteras[i-1];
	}
	
	int idx = blockIdx.x + blockDim.x + threadIdx.x;
	int idy = blockIdx.y + blockDim.y + threadIdx.y;
	
	mat2[idx*width + idy] = mat[idx*width + idy] - (diverg[idx] + diverg[idy])/(width-2);
}

int main()
{
	int N; 											// number of elements (the same as the width of the matrix).
	int numblocks;									// number of necessary blocks in the GPU
	int b; 											// dimension of the block
	float* M;										// matrix of distances
	float* Mt;										// temporal matrix for finding the smallest values.
	float* r;										// array of divergences
	
	char buffer[100];
	try{
		printf("Name of the input_file: ");
		scanf("%s",buffer);
		ifstream input(buffer);
		input>>N;									// getting the number of elements.
		printf("%d elements.\n",N);
		
		// garbage
		input.getline(buffer,100);
		input.getline(buffer,100);
		input.getline(buffer,100);
		
		M = new float[N*N];
		r = new float[N];
		
		// Initialize the matrix with 0-values
		for(int i=0; i<N; i++)
			for(int j=0; j<N; j++)
				M[i*N+j]=0;
		
		// Passing data from input to Matrix
		for(int i=1; i<N; i++)
			for(int j=0; j<i; j++)
				input>>M[i*N + j];
				
		// Printing Matrix
		printf("Printing input matrix");
		for(int i=0; i<N; i++){
			for (int j=0; j<N; j++)
				printf("%4.2f ",M[i*N + j]);
			printf("\n");
		}
		printf("----------------------- o ----------------------\n\n");
				
		input.close();
	}catch(exception& e){
		printf("Problem trying to read file.\n");
		return 1;
	}
	
	while(N>2)
	{
		printf("***********************N=%d***********************\n\n",N);
		
		numblocks = ceil((float)N/MAX_THREADS);			// Update the number of blocks for every iteration.
		r = new float[N];
		
		Mt = new float[N*N];							// initializing the temporal Matrix.
		
		float* r_d;										// Allocate divergency array in the device.
		hipMalloc((void**) &r_d, sizeof(float)*N);
		
		float* M_d;										// Allocate distance matrix in the device and copy.
		hipMalloc((void**) &M_d, sizeof(float)*N*N);
		hipMemcpy(M_d,M,sizeof(float)*N*N,hipMemcpyHostToDevice);
		
		nj_step1<<<numblocks,N>>>(M_d,r_d,N);			// Kernel launch.
		
		hipMemcpy(r,r_d,sizeof(float)*N,hipMemcpyDeviceToHost);	// Copying response array to the Host.
		
		// Printing new divergence matrix.
		for(int i=0; i<N; i++)
			printf("%4.2f ",r[i]);
		printf("\n");
		//int numblocks = ((N/b)*((N/b)+1))/2.0;
		//float* matrix_temp;
		
		//int* fronteras = new int[N/b];
		//for(int i=0; i<N/b ; i++)
		//{
	//		fronteras[i] = ((i+1)*(i+2))/2.0;
	//	}
		
		//nj_step2<<<numblocks,(b,b)>>>(matrix,matrix_temp,divergencias,width,fronteras);
		N = N - 1;
	}
	return 0;
}