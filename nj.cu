
#include <hip/hip_runtime.h>
#include <math.h>
#include <fstream>
#include <stdio.h>
#include <exception>

#define MAX_THREADS 1024
#define INF 		99999999999.0
#define PI			3.14159265

using namespace std;


// 3 points.
__global__ void nj_step1(float* mat, float* res,int width)		// Calculate the tree-divergence for every object.
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < width){
		float rpta = 0.0f;
		for(int i=0; i<width; i++){
			if(i<idx)
				rpta += mat[idx*width + i];
			else
				rpta += mat[i*width + idx];
		}
		res[idx] = rpta;
	}
}

// 6 points.
__global__ void nj_step2(float* mat_t, float* mat, float* diverg, int width, int* limits) // Calculate a new matrix (Mt) of distances.
{
	int bx = blockIdx.x;
	
	int k = 0;
	int blockfil = 0;
	int blockcol = 0;
	while(limits[k] != NULL && limits[k] < bx){
		k++;
	}
	
	if(k!=0)
		blockfil = k - 1;
	
	if(k!=0)
		blockcol = bx - limits[k - 1] - 1;
	
	int idx = threadIdx.x;
	int idy = threadIdx.y;
	
	if( (limits[k]) == blockcol){
		int i = (blockfil * blockDim.x) + idx;
		int j = (blockcol * blockDim.y) + idy;
		
		if (i < width && j < width){
			if(idy < idx){
				mat_t[i*width + j] = mat[i*width + j] - (diverg[i] + diverg[j])/(width-2);
			}else
				mat_t[i*width + j] = PI;
		}
	}else{
		int i = (blockfil * blockDim.x) + idx;
		int j = (blockcol * blockDim.y) + idy;
		
		if (i < width && j < width)
			mat_t[i*width + j] = mat[i*width + j] - (diverg[i] + diverg[j])/(width-2);
	}
}

int main()
{
	int N; 											// number of elements (the same as the width of the matrix).
	int numblocks;									// number of necessary blocks in the GPU
	int b = 8; 										// dimension of the block (blocks of 8x8 is 64 threads in the block, 
													// which benefits CUDA 'cause is it multiple of 32 (for warp control)
	float* M;										// matrix of distances
	float* Mt;										// temporal matrix for finding the smallest values.
	float* r;										// array of divergences
	
	char buffer[100];
	try{
		printf("Name of the input_file: ");
		scanf("%s",buffer);
		ifstream input(buffer);
		input>>N;									// getting the number of elements.
		printf("%d elements.\n",N);
		
		// garbage
		input.getline(buffer,100);
		input.getline(buffer,100);
		input.getline(buffer,100);
		
		M = new float[N*N];
		r = new float[N];
		
		// Initialize the matrix with 0-values
		for(int i=0; i<N; i++)
			for(int j=0; j<N; j++)
				M[i*N+j]=0;
		
		// Passing data from input to Matrix
		for(int i=1; i<N; i++)
			for(int j=0; j<i; j++)
				input>>M[i*N + j];
				
		// Printing Matrix
		printf("Printing input matrix");
		for(int i=0; i<N; i++){
			for (int j=0; j<N; j++)
				printf("%4.2f ",M[i*N + j]);
			printf("\n");
		}
		printf("----------------------- o ----------------------\n\n");
				
		input.close();
	}catch(exception& e){
		printf("Problem trying to read file.\n");
		return 1;
	}
	
	while(N>2)
	{
		printf("***********************N=%d***********************\n\n",N);
		
		numblocks = ceil((float)N/MAX_THREADS);			// Update the number of blocks for every iteration.
		r = new float[N];
		
		Mt = new float[N*N];							// initializing the temporal Matrix.
		
		float* r_d;										// Allocate divergency array in the device.
		hipMalloc((void**) &r_d, sizeof(float)*N);
		
		float* M_d;										// Allocate distance matrix in the device and copy.
		hipMalloc((void**) &M_d, sizeof(float)*N*N);
		hipMemcpy(M_d,M,sizeof(float)*N*N,hipMemcpyHostToDevice);
		
		nj_step1<<<numblocks,N>>>(M_d,r_d,N);			// Kernel launch for step 1.
		
		hipMemcpy(r,r_d,sizeof(float)*N,hipMemcpyDeviceToHost);	// Copying response array to the Host.
		
		// Printing new divergence matrix.
		for(int i=0; i<N; i++)
			printf("%4.2f ",r[i]);
		printf("\n");
		
		int nb = ceil((double)((double)N/(double)b));
		
		printf("nb: %d\n",nb);
		
		int numblocks = (nb*(nb+1))/2.0;			// Number of blocks like a triangular matrix.
		
		printf("number of blocks for step2: %d\n\n",numblocks);
		
		int* limits = new int[nb];
		for(int i=0; i<nb ; i++)
			limits[i] = (int)((((i+1)*(i+2))/2.0) - 1);
		
		float* Mt_d;
		hipMalloc((void**) &Mt_d, sizeof(float)*N*N);
		int* limits_d;
		hipMalloc((void**) &limits_d, sizeof(int)*nb);
		hipMemcpy(limits_d,limits,sizeof(int)*nb,hipMemcpyHostToDevice);
		
		nj_step2<<<numblocks,dim3(b,b)>>>(Mt_d,M_d,r_d,N,limits_d);	// Kernel launch for step 2.
		
		hipMemcpy(Mt,Mt_d,sizeof(float)*N*N,hipMemcpyDeviceToHost);	// Copying response matrix to the Host.
		
		// Printing temporal distance matrix (Mt).
		
		printf("Printing temporal distance matrix (Mt).\n");
		for(int i=0; i<N; i++){
			for(int j=0; j<N; j++)
				printf("%4.2f ",Mt[i*N + j]);
			printf("\n");
		}
		scanf("%s",buffer);
		
		// Step 3: Select objects "i" and "j" where M[i][j] is the minimum. 1 point.
		// Step 4: Create a new object U and delete "i" and "j". 3 points.
		// Step 5: Calculate distances from "i" to U and "j" to U. 2 points.
		// Step 6: Calculate the distance between U and the rest. 4 points.
		
		N = N - 1;
	}
	return 0;
}